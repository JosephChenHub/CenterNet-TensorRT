#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include "dcn_v2.hpp"
#include "dcn_v2_im2col_cuda.h"
#include <stdio.h>
#include <vector>

#define MAX_BATCH_SIZE 32
//#define USE_D_ONES

#define CHECK_CUDA(e) { if(e != hipSuccess) { \
    printf("cuda failure: %s:%d: '%s'\n", __FILE__, __LINE__, \
            hipGetErrorString(e)); \
    exit(0); \
} \
}

#define CHECK_LAST_ERR(func) { \
    hipError_t e = hipGetLastError();\
    if (e != hipSuccess) {\
        printf("cuda failure of %s: %s:%d: '%s'\n", func, __FILE__, __LINE__, \
                hipGetErrorString(e)); \
        exit(-1); \
    } \
}

/// Static class fields initialization
nvinfer1::PluginFieldCollection DCNPluginCreator::_mFC{};
std::vector<nvinfer1::PluginField> DCNPluginCreator::_mPluginAttributes;

const char* cublasGetErrorString(hipblasStatus_t status) {
    switch(status) {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

hipblasHandle_t blas_handle() {
    static int init[16] = {0};
    static hipblasHandle_t handle[16];
    const int n = 0;
    //hipError_t status = hipGetDevice(&n);
    if(!init[n]) {
        hipblasStatus_t st = hipblasCreate(&handle[n]);
        if (st != HIPBLAS_STATUS_SUCCESS) {
            printf("blas_handle create failed! %s:%d, code:%s\n", __FILE__, __LINE__, cublasGetErrorString(st));
        }
        init[n] = 1;
    }
    return handle[n];
}


// bias: (k,), output: (batch_num, k, n)
__global__ void ones_mul_bias(const float* bias, float* output, 
        const int n, const int k, const int batch_num) {
    const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= batch_num * n * k) return;

    const size_t col_id = (tid % (n * k)) / n;

    output[tid] = bias[col_id];
} 

template <typename T>
void createBatchBuffers(T* buff[], T* data, const size_t len_per_batch, const int batch_num) {
    for(int i = 0; i < batch_num; ++i) {
        buff[i] = data + len_per_batch * i;
    }
}


__inline__ size_t divUp(size_t num, int threads) {
    return (num + threads - 1) / threads;
}

int DCNPlugin::enqueue(int batchSize, const void *const *inputs, void **outputs,
        void *workspace, hipStream_t stream) {

    /// input' shape is CHW
    const int in_channels = this->_inputDims.d[0];
    const int input_height = this->_inputDims.d[1];
    const int input_width = this->_inputDims.d[2];

    const int out_channels = this->_outputDims.d[0];
    const int output_height = this->_outputDims.d[1];
    const int output_width = this->_outputDims.d[2];

    ///
    const float* input = static_cast<const float*>(inputs[0]);
    const float* offset = static_cast<const float*>(inputs[1]);
    const float* mask = static_cast<const float*>(inputs[2]);
    const float* weight = static_cast<const float*>(inputs[3]);
    const float* bias = static_cast<const float*>(inputs[4]);
    float * output = static_cast<float *>(outputs[0]);
    hipblasHandle_t handle = blas_handle();
    hipblasSetStream(handle, stream);

    float alpha, beta;
    size_t m, n, k;
    m = out_channels;
    n = output_height * output_width;
    k = 1;
    alpha = 1.0;
    beta = 0.0;
    hipblasStatus_t st;
    //assert(batchSize==1);
    assert(batchSize <= MAX_BATCH_SIZE);

#ifdef USE_D_ONES    
    st = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,&alpha,
            _d_ones, k, bias, k,&beta, output, n);
    if (st != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm error occurred! %s : %d, error_code:%s, n:%d, m:%d, k:%d\n", __FILE__, __LINE__, 
                cublasGetErrorString(st), n, m, k);
        exit(-1);
    }
#else 
    size_t num_blocks = divUp(n * m * batchSize, 512);
    ones_mul_bias<<<num_blocks, 512, 0, stream>>>(bias, output, n, m, batchSize);
#endif 
    // im2col (offset and mask)
    /// offset: (batch_num, o_c, in_h, in_w)
    /// mask:(batch_num, o_m, in_h, in_w)
    modulated_deformable_im2col_cuda(stream,input,offset,mask, \
            batchSize, in_channels, input_height, input_width,
            output_height, output_width, _kernel_size, _kernel_size,
            _padding, _padding, _stride, _stride, _dilation, _dilation,
            _deformable_groups, _d_columns);


    m = out_channels;
    n = output_height * output_width;
    k = in_channels * _kernel_size * _kernel_size;
    alpha = 1.0;
    beta = 1.0;
    // im2col conv
    /// _d_columns: batch* (k=in_c*ker*ker) * (n=o_h*o_w)
    /// weight:  m(o_c) * k
    /// output: batch*m*n
    /// output = weight x _d_columns

    // C^T = (AB)^T = B^T A^T
    // B_COL, A_ROW, B_ROW, d_B, B_COL, d_A, A_COL, d_C, B_COL
    //if (batchSize == 1) { 
    //    st = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k,&alpha, _d_columns, n, weight, k,&beta, output, n);
    //} else 
    { 
        float* col_buff[MAX_BATCH_SIZE];
        const float* w_buff[MAX_BATCH_SIZE];
        float* o_buff[MAX_BATCH_SIZE];
        createBatchBuffers<float>(col_buff, _d_columns, n * k, batchSize);
        createBatchBuffers<float>(o_buff, output, n*m, batchSize);
        for(int i = 0; i < batchSize; ++i) w_buff[i] = weight;
        
        float** d_col_buff, **d_w_buff, **d_o_buff;
        hipMalloc(&d_col_buff, sizeof(float*) * batchSize);
        hipMalloc(&d_w_buff, sizeof(float*) * batchSize);
        hipMalloc(&d_o_buff, sizeof(float*) * batchSize);
        hipMemcpy(d_col_buff, col_buff, sizeof(float*) * batchSize, hipMemcpyHostToDevice);
        hipMemcpy(d_w_buff, w_buff, sizeof(float*) * batchSize, hipMemcpyHostToDevice);
        hipMemcpy(d_o_buff, o_buff, sizeof(float*) * batchSize, hipMemcpyHostToDevice);

        st = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
            n, m, k, &alpha, d_col_buff, n, 
            d_w_buff, k, &beta, d_o_buff, n, batchSize);

    }
    if (st != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemm error occurred! %s : %d, error_code:%s, n:%d, m:%d, k:%d\n", __FILE__, __LINE__, 
                cublasGetErrorString(st), n, m, k);
        exit(-1);
    }
    

    return 0;
}

int DCNPlugin::initialize()  {
    //std::cout << "**** initialize called! **** id:"  << this<<  std::endl;
    if(_initialized) return 0;
    auto _out_dims = this->getOutputDimensions(0, &this->getInputDims(0), 5);
    size_t ones_size = _out_dims.d[1]*_out_dims.d[2];
#ifdef USE_D_ONES    
    CHECK_CUDA(hipMalloc((void**)&_d_ones, ones_size*sizeof(float) ));      
    float* cpu_ones = new float[ones_size];
    for(size_t i = 0; i < ones_size; ++i) cpu_ones[i] = 1.0;
    CHECK_CUDA(hipMemcpy(_d_ones, cpu_ones, ones_size*sizeof(float), hipMemcpyHostToDevice));
#endif     
    size_t column_size = _inputDims.d[0] * _kernel_size * _kernel_size * ones_size * MAX_BATCH_SIZE;
    CHECK_CUDA(hipMalloc((void**)&_d_columns, sizeof(float) * column_size));

#ifdef USE_D_ONES
    delete [] cpu_ones;
#endif 
    _initialized = true;

    return 0;
}

void DCNPlugin::terminate()  {
    if(!_initialized) return;
#ifdef USE_D_ONES    
    CHECK_CUDA(hipFree(_d_ones));
#endif     
    CHECK_CUDA(hipFree(_d_columns));

    _initialized = false;
}
